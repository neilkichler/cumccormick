#include "hip/hip_runtime.h"
#include <cumccormick/arithmetic/basic.cuh>

#include "tests_common.h"

#include <stdio.h>

__device__ void print(mc<double> x)
{
    printf("(cv: %.15g, cc: %.15g, box: [%g, %g])\n", x.cv, x.cc, x.box.lb, x.box.ub);
};

__global__ void basic_kernel()
{
    mc<double> a { .cv = 1.0, .cc = 2.0, .box = { .lb = 0.0, .ub = 3.0 } };
    mc<double> b { .cv = 3.0, .cc = 4.0, .box = { .lb = 2.0, .ub = 5.0 } };

    print(a);
    print(b);

    auto c = add(a, b);
    print(c);
    auto d = sub(a, b);
    print(d);
    auto e = mul(2.0, a);
    print(e);
    auto f = sqr((a + b) - a);
    print(f);
    auto g = div(f, 2.0);
    print(g);
    auto h = exp(a - 1.5);
    print(h);
    auto i = sqrt(a + b);
    print(i);
    auto j = cos(i);
    print(j);
    auto k = sin(j);
    print(k);
    auto l = log(a);
    print(l);
    auto m = log(b);
    print(m);
    auto n = mul(a, b);
    print(n);
    auto o = recip(a);
    print(o);
    auto p = div(a, b);
    print(p);
    auto q = pown(a, 3);
    print(q);
    auto r = pown(b, 3);
    print(r);
    auto s = pown(a - 2.0, 3);
    print(s);
    auto t = abs(a - 2.0);
    print(t);
    auto u = abs(b);
    print(u);
    auto v = max(a, b);
    print(v);
    auto w = min(a, b);
    print(w);
}

__global__ void test_pown()
{
    mc<double> a { .cv = 1.0, .cc = 2.0, .box = { .lb = 0.0, .ub = 3.0 } };

    auto c = pow(a, 5);
    print(c);
    auto d = pow(a - 2.0, 5);
    print(d);
    auto e = pow(a, 4);
    print(e);
    auto f = pow(a - 2.0, 4);
    print(f);
    auto g = pow(a - 4.0, 4);
    print(g);
    auto h = pow(a + 0.0001, -4);
    print(h);
    auto i = pow(a + 1.0, -4);
    print(i);
    auto j = pow(a - 4.0, -4);
    print(j);
    auto k = pow(a - 1.5, -4);
    print(k);
    auto l = pow(a + 2.0, -5);
    print(l);
    auto m = pow(a, -5);
    print(m);
    auto n = pow(a - 4.0, -5);
    print(n);
    auto o = pow(a + 0.0001, -5);
    print(o);
}

template<typename T>
__device__ bool within_ulps(T x, T y, std::size_t n)
{
    if (x == y) {
        return true;
    }

    for (int i = 0; i < n; ++i) {
        x = std::nextafter(x, y);

        if (x == y) {
            return true;
        }
    }

    return false;
}

__device__ auto ackley(auto x, auto y)
{
    using std::numbers::e;
    using std::numbers::pi;
    return -20.0 * exp(-0.2 * sqrt(0.5 * (x * x + y * y)))
        - exp(0.5 * (cos(2.0 * pi * x) + cos(2.0 * pi * y))) + e + 20.0;
}

template<typename T>
__global__ void contains_samples_check_univariate(mc<T> *xs, int n_x, std::integral auto n)
{
    // Check that a range of samples are all contained in the mccormick bound
    int i = blockIdx.x;
    int j = threadIdx.x;

    auto contains = [](mc<T> x, T y) {
        if (!(x.cv <= y && y <= x.cc)) {
            printf("[E] Invalid bounds: x.cv = %.15g, y = %.15g, x.cc = %.15g\n", x.cv, y, x.cc);
            printf("[E] Invalid bounds: x.cv = %a, y = %a, x.cc = %a\n", x.cv, y, x.cc);
        }
        return x.cv <= y && y <= x.cc;
    };

    if (i < n) {
        mc<T> x    = xs[j];
        T x_sample = x.cv + static_cast<T>(i) * (x.cc - x.cv) / static_cast<T>(n);
        assert(contains(pow(x, 1), x_sample));
        assert(contains(pow(x, 2), pow(x_sample, 2)));
        assert(contains(pow(x, 3), pow(x_sample, 3)));
        assert(contains(pow(x, 4), pow(x_sample, 4)));
        assert(contains(pow(x, 5), pow(x_sample, 5)));
        assert(contains(abs(x), abs(x_sample)));
        assert(contains(exp(x), exp(x_sample)));
        assert(contains(fabs(x), fabs(x_sample)));
        assert(contains(neg(x), -x_sample));
        assert(contains(sqr(x), pow(x_sample, 2)));
        assert(contains(cos(x), cos(x_sample)));
        // assert(contains(sin(x), sin(x_sample)));

        if (inf(x) >= 0) {
            assert(contains(log(x), log(x_sample)));
            assert(contains(recip(x), pow(x_sample, -1)));
            assert(contains(sqrt(x), sqrt(x_sample)));
        }
    }
}

template<typename T>
__global__ void contains_samples_check_bivariate(mc<T> *xs, mc<T> *ys, std::integral auto n)
{
    // Check that a range of samples are all contained in the mccormick bound
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k = threadIdx.y;

    auto contains = [](mc<T> x, T y) {
        return x.cv <= y && y <= x.cc;
    };

    if (i < n) {
        mc<T> x    = xs[j];
        mc<T> y    = ys[k];

        T x_sample = x.cv + static_cast<T>(i) * (x.cc - x.cv) / static_cast<T>(n);
        T y_sample = y.cv + static_cast<T>(i) * (y.cc - y.cv) / static_cast<T>(n);
        assert(contains(x + y, x_sample + y_sample));
        assert(contains(x - y, x_sample - y_sample));
        assert(contains(x * y, x_sample * y_sample));
        assert(contains(max(x, y), max(x_sample, y_sample)));
        assert(contains(min(x, y), min(x_sample, y_sample)));
    }
}

__global__ void test_fn_kernel()
{
    mc<double> x { .cv = 1.5, .cc = 1.5, .box = { .lb = 1.0, .ub = 2.0 } };
    mc<double> y { .cv = 0.6, .cc = 0.65, .box = { .lb = 0.5, .ub = 0.7 } };
    mc<double> z { .cv = 0.2, .cc = 1.0, .box = { .lb = -1.0, .ub = 2.0 } };

    auto xy = x * y;
    assert(within_ulps(xy.cv, 0.85, 1));
    assert(within_ulps(xy.cc, 1.0, 1));

    auto xz = x * z;
    assert(within_ulps(xz.cv, -0.3, 1));
    assert(within_ulps(xz.cc, 2.0, 1));

    auto yz = y * z;
    assert(within_ulps(yz.cc, 0.8, 1));

    auto xexp = x * exp(-pow(x, 2));
    assert(within_ulps(xexp.cv, 0x1.75bb077991bc3p-4, 1));
    assert(within_ulps(xexp.cc, 0x1.9fea64b7c3615p-2, 1));

    // auto sincospow = sin(pown(y, -3)) * cos(pown(y, 2)); // -9.358968236779348e-01, 6.095699354841704e-01
    // printf("sincospow.cv: %a, %.15f\n", sincospow.cv, sincospow.cv);
    // printf("sincospow.cc: %a, %.15f\n", sincospow.cc, sincospow.cc);

    auto ack = ackley(x, y);
    // printf("ack.cv: %a, %.15f\n", ack.cv, ack.cv);
    // printf("ack.cc: %a, %.15f\n", ack.cc, ack.cc);
}

void bounds_kernel(hipStream_t stream)
{
    constexpr int n_samples = 512;
    constexpr int n_xs      = 8;

    mc<double> xs[n_xs] = {
        { .cv = 0.6, .cc = 0.65, .box = { .lb = 0.0, .ub = 0.7 } },
        { .cv = 7.6, .cc = 7.65, .box = { .lb = 6.1, .ub = 7.7 } },
        { .cv = 50.6, .cc = 100.65, .box = { .lb = 50.0, .ub = 100.7 } },
        { .cv = 3.6, .cc = 3.85, .box = { .lb = -4.1, .ub = 7.7 } },
        { .cv = -0.01, .cc = 0.01, .box = { .lb = -0.1, .ub = 0.1 } },
        { .cv = -0.01, .cc = 0.01, .box = { .lb = -0.01, .ub = 0.01 } },
        { .cv = 10000.01, .cc = 10001.01, .box = { .lb = 0.0, .ub = 100000.0 } },
        { .cv = -3.96, .cc = -3.25, .box = { .lb = -4.1, .ub = -3.1 } },
    };

    mc<double> *d_xs;
    CUDA_CHECK(hipMalloc(&d_xs, n_xs * sizeof(mc<double>)));
    CUDA_CHECK(hipMemcpy(d_xs, xs, n_xs * sizeof(mc<double>), hipMemcpyHostToDevice));

    contains_samples_check_univariate<<<n_samples, n_xs>>>(d_xs, n_xs, n_samples);

    mc<double> *d_ys;
    constexpr int n_ys  = 2;
    mc<double> ys[n_ys] = {
        { .cv = -0.5, .cc = 0.5, .box = { .lb = -1.0, .ub = 3.0 } },
        { .cv = 0.5, .cc = 2.5, .box = { .lb = 0.0, .ub = 3.0 } },
    };

    CUDA_CHECK(hipMalloc(&d_ys, n_ys * sizeof(mc<double>)));
    CUDA_CHECK(hipMemcpy(d_ys, ys, n_ys * sizeof(mc<double>), hipMemcpyHostToDevice));

    dim3 blocks(n_xs, n_ys);
    contains_samples_check_bivariate<<<n_samples, blocks>>>(d_xs, d_ys, n_samples);

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
}

void basic_kernel(hipStream_t stream)
{
    basic_kernel<<<1, 1, 0, stream>>>();
}

void pown_kernel(hipStream_t stream)
{
    test_pown<<<1, 1, 0, stream>>>();
}

void fn_kernel(hipStream_t stream)
{
    test_fn_kernel<<<1, 1, 0, stream>>>();
}
