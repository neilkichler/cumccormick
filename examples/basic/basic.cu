#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cumccormick/arithmetic/basic.cuh>

#include "../common.h"
#include "../tests/tests_common.h"

__device__ auto beale(auto x, auto y)
{
    return pow(1.5 - x * (1 - y), 2)
        + pow(2.25 - x * (1 - sqr(y)), 2)
        + pow(2.625 - x * (1 - pow(y, 3)), 2);
}

__global__ void kernel(auto *xs, auto *ys, auto *res, std::integral auto n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        res[i] = beale(xs[i], ys[i]);
    }
}

int main()
{
    constexpr int n = 256;
    using T = mc<double>;
    T xs[n], ys[n], res[n];

    // generate dummy data
    for (int i = 0; i < n; i++) {
        double v = i;
        xs[i] = { .cv = -v, .cc = v, .box = { .lb = -v, .ub = v } };
        ys[i] = { .cv = -v, .cc = v, .box = { .lb = -v, .ub = v } };
    }

    mc<double> *d_xs, *d_ys, *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_ys, n * sizeof(*ys)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ys, ys, n * sizeof(*ys), hipMemcpyHostToDevice));

    kernel<<<n, 1>>>(d_xs, d_ys, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto r = res[0];
    printf("beale(0, 0) = " MCCORMICK_FORMAT, r.box.lb, r.cv, r.cc, r.box.ub);

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
