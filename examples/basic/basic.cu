#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../common.h"
#include "../tests/tests_common.h"

__device__ auto beale(auto x, auto y)
{
    return pow(1.5 - x * (1 - y), 2)
        + pow(2.25 - x * (1 - sqr(y)), 2)
        + pow(2.625 - x * (1 - pow(y, 3)), 2);
}

__global__ void kernel(auto *xs, auto *ys, auto *res, std::integral auto n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        res[i] = beale(xs[i], ys[i]);
    }
}

int main()
{
    constexpr int n = 256;
    using T = mc<double>;
    T xs[n], ys[n], res[n];

    // generate dummy data
    for (int i = 0; i < n; i++) {
        double v = i;
        xs[i] = {{ .lb = -v, .cv = -v, .cc = v, .ub = v }};
        ys[i] = {{ .lb = -v, .cv = -v, .cc = v, .ub = v }};
    }

    mc<double> *d_xs, *d_ys, *d_res;
    CUDA_CHECK(hipMalloc(&d_xs, n * sizeof(*xs)));
    CUDA_CHECK(hipMalloc(&d_ys, n * sizeof(*ys)));
    CUDA_CHECK(hipMalloc(&d_res, n * sizeof(*res)));

    CUDA_CHECK(hipMemcpy(d_xs, xs, n * sizeof(*xs), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ys, ys, n * sizeof(*ys), hipMemcpyHostToDevice));

    kernel<<<n, 1>>>(d_xs, d_ys, d_res, n);

    CUDA_CHECK(hipMemcpy(res, d_res, n * sizeof(*res), hipMemcpyDeviceToHost));

    auto r = res[0];
    printf("beale(0, 0) = " MCCORMICK_FORMAT "\n", r.box.lb, r.cv, r.cc, r.box.ub);

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_res));

    return 0;
}
