#include "hip/hip_runtime.h"
#include <cumccormick/arithmetic/basic.cuh>

#include <cstdio>
#include <span>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../common.h"
#include "../tests/tests_common.h"

#define fn     __device__ auto
#define lambda [] __device__

fn rosenbrock(auto x, auto y)
{
    double a = 1.0;
    double b = 100.0;
    return pow(a - x, 2) + b * pow((y - pow(x, 2)), 2);
}

fn model(auto x, auto y)
{
    auto rosen = rosenbrock(x, y);
    auto z     = cos(rosen) - x + x;
    z          = 10.0 * z;
    return z;
}

__global__ void generic_kernel(auto &&f, mc<double> *xs, mc<double> *ys, mc<double> *res, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        res[i] = f(xs[i], ys[i]);
    }
}

template<typename T>
void launch(auto &&user_kernel, std::span<mc<T>> xs, std::span<mc<T>> ys)
{
    mc<T> *d_xs;
    mc<T> *d_ys;
    mc<T> *d_res;

    const int n        = xs.size();
    const int xs_size  = xs.size_bytes();
    const int ys_size  = ys.size_bytes();
    const int res_size = xs_size;

    CUDA_CHECK(hipMalloc(&d_xs, xs_size));
    CUDA_CHECK(hipMalloc(&d_ys, ys_size));
    CUDA_CHECK(hipMalloc(&d_res, res_size));

    CUDA_CHECK(hipMemcpy(d_xs, xs.data(), xs_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_ys, ys.data(), ys_size, hipMemcpyHostToDevice));

    // TODO: have a global context from which we can make use of the
    // gpu configuration, stream, threadblocks etc.

    // TODO: make kernel variadic in input arguments
    generic_kernel<<<128, 1>>>(user_kernel, d_xs, d_ys, d_res, n);

    std::vector<mc<T>> res(n);
    CUDA_CHECK(hipMemcpy(res.data(), d_res, res_size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_xs));
    CUDA_CHECK(hipFree(d_ys));
    CUDA_CHECK(hipFree(d_res));

    printf("Results: \n");
    for (auto r : res) {
        printf(MCCORMICK_FORMAT "\n", r.box.lb, r.cv, r.cc, r.box.ub);
    }
}

void user_kernel_multiple_inputs(cuda_ctx ctx)
{
    std::vector<mc<double>> xs {
        { .cv = -1.96, .cc = 1.25, .box = { .lb = -2.0, .ub = 2.0 } },
        { .cv = 0.6, .cc = 0.65, .box = { .lb = 0.0, .ub = 0.7 } },
        { .cv = 7.6, .cc = 7.65, .box = { .lb = 6.1, .ub = 7.7 } },
        { .cv = 50.6, .cc = 100.65, .box = { .lb = 50.0, .ub = 100.7 } },
        { .cv = 3.6, .cc = 3.85, .box = { .lb = -4.1, .ub = 7.7 } },
        { .cv = -0.01, .cc = 0.01, .box = { .lb = -0.1, .ub = 0.1 } },
        { .cv = -0.01, .cc = 0.01, .box = { .lb = -0.01, .ub = 0.01 } },
        { .cv = 10000.01, .cc = 10001.01, .box = { .lb = 0.0, .ub = 100000.0 } },
        { .cv = -3.96, .cc = -3.25, .box = { .lb = -4.1, .ub = -3.1 } },
    };

    std::vector<mc<double>> ys {
        { .cv = -0.5, .cc = 0.5, .box = { .lb = -1.0, .ub = 3.0 } },
        { .cv = 0.5, .cc = 2.5, .box = { .lb = 0.0, .ub = 3.0 } },
        { .cv = -0.5, .cc = 0.5, .box = { .lb = -1.0, .ub = 3.0 } },
        { .cv = 0.5, .cc = 2.5, .box = { .lb = 0.0, .ub = 3.0 } },
        { .cv = -0.5, .cc = 0.5, .box = { .lb = -1.0, .ub = 3.0 } },
        { .cv = 0.5, .cc = 2.5, .box = { .lb = 0.0, .ub = 3.0 } },
        { .cv = -0.5, .cc = 0.5, .box = { .lb = -1.0, .ub = 3.0 } },
        { .cv = 0.5, .cc = 2.5, .box = { .lb = 0.0, .ub = 3.0 } },
        { .cv = -0.5, .cc = 0.5, .box = { .lb = -1.0, .ub = 3.0 } },
    };

    //
    // Using a lambda function as the user kernel
    //
    auto user_kernel = lambda(auto x, auto y)
    {
        return pow(1.0 - x, 2) + 100.0 * pow((y - pow(x, 2)), 2);
    };

    launch<double>(user_kernel, xs, ys);

    //
    // Using a predefined function (must be wrapped in a lambda)
    //
    launch<double>([] __device__(auto x, auto y) { return model(x, y); }, xs, ys);
}

int main()
{
    CUDA_CHECK(hipSetDevice(0));

    std::array<cuda_buffer, n_streams> buffers {};

    std::array<hipStream_t, n_streams> streams {};
    for (auto &stream : streams)
        CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::array<hipEvent_t, n_streams> events {};
    for (auto &event : events)
        CUDA_CHECK(hipEventCreateWithFlags(&event, hipEventDisableTiming));

    cuda_ctx ctx { buffers, streams, events };
    user_kernel_multiple_inputs(ctx);

    for (auto &event : events)
        CUDA_CHECK(hipEventDestroy(event));

    for (auto &stream : streams)
        CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
    return 0;
}
